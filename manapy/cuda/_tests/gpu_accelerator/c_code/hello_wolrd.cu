
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel() 
{ 
  printf("Hello, world from the device!\n"); 
} 

int main() 
{ 
  myKernel<<<1,10>>>(); 
  hipDeviceSynchronize();
} 